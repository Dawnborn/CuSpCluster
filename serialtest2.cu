#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <string>
#include <fstream>

#include ""
#include "./src/utils.h"

constexpr int GRID_SIZE = 1024;
constexpr int BLOCK_SIZE = 512;
constexpr int WARP_SIZE = 32;

constexpr int kSrcCount = 30;
constexpr int kCoords = 10;
constexpr int kClusterCount = 6;
const int loop_iteration = 300;

const std::string path = "./../../data/blob/";

enum cluster_kernel_t {cpu,naive,SharedMemory,ParallelReduction,MoreParallelReduction};
enum call_func_t {};

template<typename ValueType, typename IndexType>
int read_csr(ValueType *cv, IndexType *rp, IndexType *ci){
    std::ifstream f1((path+"tcsr_values.txt"), std::ifstream::in);
    std::ifstream f2((path+"tcsr_col.txt"), std::ifstream::in);
    std::ifstream f3((path+"tcsr_ptr.txt"), std::ifstream::in);
    int count;
    ValueType value;
    IndexType idx;

    if(f1){
        for(count = 0;f1>>value; count++){
            cv[count] = value;
        }
        f1.close(); //pin(cv,1,num_nz,__LINE__);
        //    num_nz = count;
    }
    if(f2){
        for(count = 0; f2>>idx; count++){
            ci[count] = idx;
        }
        f2.close();//pin(ci,1,num_nz,__LINE__);
    }
    else{
        std::cout<<"fail to open col_idx!"<<std::endl;
    }

    if(f3){
        for(count = 0; f3>>idx ;count++){
            rp[count] = idx;
        }
        f3.close();
    }
    else{
        std::cout<<"fail to open row_ptrs!"<<std::endl;
    }
    return 0;
}

template<typename ValueType>
int get_nz(int &nz){
    std::ifstream ff;
    ff.open((path+"tcsr_ptr.txt"), std::ifstream::in);
    int count = 0;
    ValueType tmp;
    while(ff>>tmp){
        count++;
    }
    ff.close();
    nz = tmp;
    return 0;
}

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
static inline __device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
            (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                                             __longlong_as_double(assumed)));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

template<typename ValueType, typename IndexType, int Coord, int ClusterCount, int SrcCount>
__global__ void distanceKernel1(
        ValueType* c /*[ClusterCount][SrcCount]*/,
        const ValueType* a_cv,
        const IndexType* a_ptrs,
        const IndexType* a_cid,
        const ValueType* b
){
    for(unsigned int i = 0; i*gridDim.x*blockDim.x<SrcCount;i++){ //FIXME: SrcCount-1 ?
        unsigned int iData = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int iCluster = blockIdx.y * blockDim.y + threadIdx.y;
        iData = iData + i * gridDim.x * blockDim.x;
        if(iData<SrcCount && iCluster<ClusterCount){
            c[iData+iCluster*SrcCount] = 0;
            for(unsigned int j = a_ptrs[iData]; j < a_ptrs[iData+1]; j++){
                c[iData+iCluster*SrcCount] += std::pow((b[iCluster*Coord+a_cid[j]] - a_cv[j]),2);
            }
        }
    }
}

template<typename ValueType, int ClusterCount, int SrcCount>
__global__ void membershipKernel1(
        ValueType *Dst, /*[ClusterCount][SrcCount]*/
        int *Membership,
        bool *dChanged
){
    unsigned int iData = threadIdx.x + blockDim.x * blockIdx.x;
    bool flag_changed = false;
    bool regdChanged = *dChanged;
    if(iData < SrcCount){
        int min_cluster = 0;
        int pre_cluster = Membership[iData];
        ValueType min_dist = Dst[SrcCount*min_cluster + iData];
        for(unsigned int iCluster = 1; iCluster < ClusterCount; iCluster++){
            if(Dst[SrcCount*iCluster + iData] < min_dist){
                min_cluster = iCluster;
                min_dist = Dst[SrcCount*iCluster + iData];
            }
        }
        if(min_cluster!=pre_cluster) {
            Membership[iData] = min_cluster;
            flag_changed  = true;
        }
        if(flag_changed&&(regdChanged==false)){
            *dChanged = true;
        }
    }
}

template<typename ValueType, typename IndexType, int Coord, int ClusterCount, int SrcCount>
inline void updateMembership(
        ValueType *dDis, /*[ClusterCount][SrcCount]*/
        int *dMembership, /*[SrcCount]*/
        bool *dChanged,
        const ValueType *dData_cv,
        const IndexType *dData_ptr,
        const IndexType *dData_cid,
        const ValueType *dClusters
){
    dim3 grid(GRID_SIZE, 16, 1), block(BLOCK_SIZE, 2, 1);
    distanceKernel1<ValueType,IndexType,Coord,ClusterCount,SrcCount><<<grid,block>>>(dDis,dData_cv,dData_ptr,dData_cid,dClusters);
    checkCuda();

    checkCuda(hipDeviceSynchronize());
//    checkCuda(hipMemcpy(Dis,dDis,ClusterCount*SrcCount*sizeof(ValueType),hipMemcpyDeviceToHost));

    dim3 grid2((SrcCount + BLOCK_SIZE - 1) / BLOCK_SIZE), block2(BLOCK_SIZE);
    membershipKernel1<ValueType,ClusterCount,SrcCount><<<grid2,block2>>>(dDis,dMembership,dChanged);
    checkCuda();
    checkCuda(hipDeviceSynchronize());
}

template<typename ValueType, typename IndexType, int Coord, int ClusterCount, int SrcCount>
void updateClustersMemberCount_cpu(
        ValueType *Clusters, /*[Coord][ClusterCount]*/
        int *MemberCount,
        const int *Membership,
        const ValueType *Data_cv,
        const IndexType *Data_ptr,
        const IndexType *Data_cid
){
    std::fill(Clusters,Clusters+Coord*ClusterCount,0);
    std::fill(MemberCount,MemberCount+ClusterCount,0);
    for(unsigned int i = 0; i < SrcCount; i++){
        MemberCount[Membership[i]] += 1;
        for(unsigned int j = Data_ptr[i]; j < Data_ptr[i+1]; j++){
            Clusters[ClusterCount*Data_cid[j]+Membership[i]] += Data_cv[j];
        }
    }

    for(unsigned int j = 0; j < ClusterCount; j++){
        if(MemberCount[j]!=0) {
            for (unsigned int idim = 0; idim < Coord; idim++) {
                Clusters[ClusterCount * idim + j] /= MemberCount[j];
            }
        }
    }
}

template<typename ValueType, typename IndexType, int Coord, int ClusterCount, int SrcCount>
void updateClusters_OMP(
        ValueType *Clusters, /*[Coord][ClusterCount]*/
        int *MemberCount,
        const int *Membership,
        const ValueType *Data_cv,
        const IndexType *Data_ptr,
        const IndexType *Data_cid
){} //TODO: OMP update Clusters


template<typename ValueType, int Length>
__global__ void fillKernel(ValueType *Array, ValueType val){
    for (unsigned int i = threadIdx.x; i < Length; i += blockDim.x) {
        Array[i] = val;
    }
}

template<int SrcCount>
__global__ void updateMemberCountKernel(const int *dMembership, int *dMemberCount) {
    /**
     * one block for one kind of cluster, load data in the shared memory and then sum them up
     * no need to initialize the MemberCount
     *
     * grid_size: ClusterCount
     * block_size: BLOCK_SIZE
     * **/

    unsigned int iCluster = blockIdx.x;
    unsigned int tid = threadIdx.x;
    __shared__ int sm[SrcCount]; //FIXME: for extreme large SrcCount?
    for (unsigned int i = tid; i < SrcCount; i += blockDim.x) {
        sm[i] = (int) (dMembership[i] == iCluster);
    }
    __syncthreads();

    /** perform parallel reduction **/
    for (unsigned int step = SrcCount / 2; step >= WARP_SIZE; step /= 2) {
        __syncthreads();
        if (tid + step < SrcCount) {
            sm[tid] += sm[tid + step];
        }
    }
    int reg_data = sm[tid];
#pragma unroll
    for (unsigned int step = WARP_SIZE / 2; step > 0; step /= 2) {
        reg_data += __shfl_xor_sync(0xffffffff, reg_data, step, WARP_SIZE);
    }

    /** write back to MemberCount **/
    if (threadIdx.x == 0) {
        if (SrcCount % 2 == 1) {
            reg_data += sm[SrcCount - 1];
        }
        dMemberCount[iCluster] = reg_data;
    }
}

template<int ClusterCount, int SrcCount>
inline void updateMemberCount(const int *dMembership, int *dMemberCount){
    dim3 grid(ClusterCount);
    dim3 block(SrcCount);
    updateMemberCountKernel<SrcCount><<<grid,block>>>(dMembership,dMemberCount);
}

template<typename ValueType, typename IndexType, int Coord, int ClusterCount, int SrcCount>
__global__ void updateClustersKernel_naive(
        ValueType *dClusters, /*[kClusterCount][kCoord]*/
        const int *dMembership, /*[kSrcCount]*/
        const int *dMemberCount,
        const ValueType *dData_cv,
        const IndexType *dData_ptr,
        const IndexType *dData_cid
) {
    /**
     * naive GPU version, one line of threads for one cluster, used as reference
     *
     * grid_size: 1
     * block_size: (kClusterCount,64)
     **/
    unsigned int iCluster = threadIdx.x;
    int regMemberNum = dMemberCount[iCluster];
    if (regMemberNum != 0) {
        for (unsigned int i = 0; i < SrcCount; i++) {
            if (dMembership[i] == iCluster) {
                for (IndexType j = dData_ptr[i] + threadIdx.y; j < dData_ptr[i + 1]; j += blockDim.y) {
                    atomicAdd(&dClusters[iCluster * Coord + dData_cid[j]], dData_cv[j] / (ValueType) regMemberNum);
                }
            }
        }
    }
}

template<typename ValueType, typename IndexType, int Coord, int ClusterCount, int SrcCount>
__global__ void updateClustersKernel_SharedMemory(
        ValueType *dClusters, /*[kClusterCount][kCoord]*/
        const int *dMembership, /*[kSrcCount]*/
        const int *dMemberCount,
        const ValueType *dData_cv,
        const IndexType *dData_ptr,
        const IndexType *dData_cid
){
    /**
     * one block update one clusters, with atomic lock write-back
     *
     * grid_size: 1
     * block_size: BLOCK_SIZE
     * **/
    __shared__ ValueType smCluster[Coord]; //here we store the datas that belong to one cluster, and reduction later
    unsigned int block_size = blockDim.x;
    unsigned int iCluster = blockIdx.x;
    int regMemberNum = dMemberCount[iCluster];
    if(regMemberNum!=0) {
        /** initialize the shared memory **/
        for (unsigned int i = threadIdx.x; i < Coord; i += block_size) {
            smCluster[i] = (ValueType) 0;
        }
        __syncthreads();

        /** sum up with atomic lock **/
        for (unsigned int i = threadIdx.x; i < SrcCount; i += block_size) {
            if (dMembership[i] == iCluster) {
                for (IndexType j = dData_ptr[i]; j < dData_ptr[i + 1]; j++) {
//                smCluster[dData_cid[j]] += dData_cv[j]/regMemberNum; //atomic add
                    atomicAdd(&smCluster[dData_cid[j]], dData_cv[j]);
                }
            }
        }
        __syncthreads();

        /** write back **/
        for (unsigned int i = threadIdx.x; i < Coord; i += block_size) {
            dClusters[iCluster * Coord + i] = smCluster[i] / (ValueType)regMemberNum;
        }
    }
}


template<typename ValueType, typename IndexType, int Coord, int ClusterCount, int SrcCount>
__global__ void updateClusterKernel_ParallelReduction1(
        ValueType *dClusters, /*[kClusterCount][kCoord]*/
        const int *dMembership, /*[kSrcCount]*/
        const int *dMemberCount,
        const ValueType *dData_cv,
        const IndexType *dData_ptr,
        const IndexType *dData_cid
){
    /**
     * one block picks up data that belongs to one cluster from certain number of data, and loads them in the shared memory
     * blockDim.y represents the cluster it belongs to
     * perform parallel reduction in the shared memory //TODO: warp shuffle
     * then add up all blocks that belong to the same cluster with atomicAdd
     *
     * grid_size:(ceil(SrcCount/BLOCK_SIZE),iCluster)
     * block_size:(BLOCK_SIZE/8,8) //TODO: 2d thread, threadIdx.y represent different dim
     * **/

    unsigned int iData = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int iCluster = blockIdx.y;
    int regMemberNum = dMemberCount[iCluster];
    if(regMemberNum!=0) {
        extern __shared__ __align__ (sizeof(ValueType)) char work[];
        ValueType *sm = reinterpret_cast<ValueType *>(work); /*[block_size][Coord]*/
        for (unsigned int i = threadIdx.x * blockDim.y + threadIdx.y; i < blockDim.x * Coord; i += blockDim.x * blockDim.y) {
            sm[i] = 0;
        }

        if (iData < SrcCount) {
            if (dMembership[iData] == iCluster) {
                for (unsigned int i = dData_ptr[iData] + threadIdx.y; i < dData_ptr[iData + 1]; i += blockDim.y) {
                    sm[iData * Coord + dData_cid[i]] = dData_cv[i];
                }
            }
        }
        __syncthreads();

        /** parallel reduction within the block, without warp shuffle **/
        for (unsigned int step = blockDim.x / 2; step >= WARP_SIZE; step /= 2) {
            __syncthreads();
            if (iData < step) {
                for (unsigned int iDim = threadIdx.y; iDim < Coord; iDim += blockDim.y) {
                    sm[Coord * iData + iDim] += sm[Coord * (iData + step) + iDim];
                }
            }
        }
        __syncthreads();

        /** write back to dClusters with atomic lock **/
        if (threadIdx.x == 0) {
            for (unsigned int iDim = threadIdx.y; iDim < Coord; iDim += blockDim.y) {
                atomicAdd(&dClusters[Coord * iCluster + iDim], sm[iDim] / (ValueType)regMemberNum);
            }
        }
    }
}


template<typename ValueType, typename IndexType, unsigned int yDim, int Coord, int ClusterCount, int SrcCount>
__global__ void updateClusterKernel_MoreParallelReduction_step1(
        ValueType *dIntermediate, /*[kClusterCount][ceil(SrcCount/BLOCK_SIZE.x)][kCoord]*/
        const int *dMembership, /*[kSrcCount]*/
        const int *dMemberCount,
        const ValueType *dData_cv,
        const IndexType *dData_ptr,
        const IndexType *dData_cid
) {
    /**
     * use intermedia variable to store the reduced result from all blocks and later perform reduction on them (in another new kernel)
     *
     * grid_size:(ceil(SrcCount/BLOCK_SIZE.x),iCluster)
     * block_size:(BLOCK_SIZE/8,8) (yDim,8)
     * **/

    unsigned int iData = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int iCluster = blockIdx.y;
    int regMemberNum = dMemberCount[iCluster];
    if (regMemberNum != 0) {
        extern __shared__ __align__ (sizeof(ValueType)) char work[];
        ValueType *sm = reinterpret_cast<ValueType *>(work); /*[block_size][Coord]*/
        for (unsigned int i = threadIdx.x * blockDim.y + threadIdx.y;
             i < blockDim.x * Coord; i += blockDim.x * blockDim.y) {
            sm[i] = 0;
        }

        if (iData < SrcCount) {
            if (dMembership[iData] == iCluster) {
                for (unsigned int i = dData_ptr[iData] + threadIdx.y; i < dData_ptr[iData + 1]; i += blockDim.y) {
                    sm[iData * Coord + dData_cid[i]] = dData_cv[i];
                }
            }
        }
        __syncthreads();

        /** parallel reduction within the block, without warp shuffle **/ //TODO：warp shuffle here
        for (unsigned int step = blockDim.x / 2; step >= WARP_SIZE; step /= 2) {
            __syncthreads();
            if (iData < step) {
                for (unsigned int iDim = threadIdx.y; iDim < Coord; iDim += blockDim.y) {
                    sm[Coord * iData + iDim] += sm[Coord * (iData + step) + iDim];
                }
            }
        }
        __syncthreads();

        /** write result in the shared memory back to the intermediate **/
        if (threadIdx.y == 0) {
            for (unsigned int i = threadIdx.x; i < Coord; i += blockDim.x) {
                dIntermediate[iCluster * yDim * Coord + blockIdx.x * Coord +
                              i] = sm[i]; //FIXME: for odd number of blocks
            }
        }
        __syncthreads();
    }
}

template<typename ValueType, typename IndexType, int Coord, int ClusterCount, int block_num>
__global__ void updateClusterKernel_MoreParallelReduction_step2(
        ValueType *dIntermediate, /*[kClusterCount][block_num][kCoord]*/
        ValueType *dClusters, /*[kClusterCount][kCoord]*/
        const int *dMemberCount
) {
    /**
     * one block for one cluster,
     *
     * grid_size: CLusterCount
     * block_size: (BLOCK_SIZE/8,8)
     * **/
    unsigned int iCluster = blockIdx.x;
    ValueType *dpIntermediate = &dIntermediate[iCluster * block_num * Coord];
    int regMemberCount = dMemberCount[iCluster];

    /** load data for reduction in the shared memory **/
    __shared__ ValueType sm[block_num * Coord];
    for (unsigned int i = threadIdx.x; i < block_num; i += blockDim.x) {
        for (unsigned int j = threadIdx.y; j < Coord; j += blockDim.y) {
            sm[Coord * i + j] = dpIntermediate[Coord * i + j];
        }
    }
    __syncthreads();

    /** perform reduction **/
    for (unsigned int step = block_num / 2; step > 0; step /= 2) {
        __syncthreads();
        if (threadIdx.x < step) {
            for (unsigned int iDim = threadIdx.y; iDim < Coord; iDim += blockDim.y) {
                sm[threadIdx.x * Coord + iDim] += dpIntermediate[(threadIdx.x + step) * Coord +
                                                                 iDim]; //FIXME: singular number of data
            }
        }
    }
    __syncthreads();

    /** write back to dClusters **/
    if (threadIdx.y == 0) {
        for (unsigned int iDim = threadIdx.x; iDim < Coord; iDim += blockDim.x) {
            dClusters[iCluster * Coord + iDim] = sm[iDim] / (ValueType) regMemberCount;
        }
    }

}

template<typename ValueType, typename IndexType, int Coord, int ClusterCount, int SrcCount>
inline void updateClusters_cuda(
        cluster_kernel_t kernel_type,
        ValueType *dClusters,
        ValueType *dIntermediate,
        const int *dMembership, /*[kSrcCount]*/
        const int *dMemberCount, /*[kClusterCount]*/
        const ValueType *dData_cv,
        const IndexType *dData_ptr,
        const IndexType *dData_cid
) {
    fillKernel<ValueType, kClusterCount * kCoords><<<1, BLOCK_SIZE>>>(dClusters, 0);
    if (kernel_type == naive) {
        dim3 block_size(ClusterCount, 64);
        updateClustersKernel_naive<ValueType, IndexType, Coord, ClusterCount, SrcCount><<<1, block_size>>>(
                dClusters,
                dMembership,
                dMemberCount,
                dData_cv,
                dData_ptr,
                dData_cid);
        checkCuda();
    } else if (kernel_type == SharedMemory) {
        dim3 block_size(BLOCK_SIZE);
        updateClustersKernel_SharedMemory<ValueType, IndexType, Coord, ClusterCount, SrcCount><<<ClusterCount, block_size>>>(
                dClusters, dMembership, dMemberCount, dData_cv, dData_ptr, dData_cid);
        checkCuda();
    } else if (kernel_type == ParallelReduction) {
        dim3 grid_size((SrcCount + BLOCK_SIZE - 1) / BLOCK_SIZE, ClusterCount);
        dim3 block_size(256);
        updateClusterKernel_ParallelReduction1<ValueType,IndexType,Coord,ClusterCount,SrcCount><<<grid_size,block_size>>>(dClusters,dMembership,dMemberCount,dData_cv,dData_ptr,dData_cid);
        checkCuda();
    } else if (kernel_type == MoreParallelReduction) {
        dim3 block_size(BLOCK_SIZE / 8, 8);
        dim3 grid_size((SrcCount + block_size.x - 1) / block_size.x, ClusterCount);
        updateClusterKernel_MoreParallelReduction_step1<ValueType, IndexType, (SrcCount + BLOCK_SIZE / 8 - 1) /
                                                                              (BLOCK_SIZE /
                                                                               8), Coord, ClusterCount, SrcCount><<<grid_size, block_size>>>(
                dIntermediate, dMembership, dMemberCount, dData_cv, dData_ptr, dData_cid);

        dim3 grid_size2(ClusterCount);
        dim3 block_size2(BLOCK_SIZE / 8, 8);
        updateClusterKernel_MoreParallelReduction_step2<ValueType, IndexType, Coord, ClusterCount,
                (SrcCount + BLOCK_SIZE / 8 - 1) / (BLOCK_SIZE / 8)><<<grid_size2, block_size2>>>(dIntermediate,
                                                                                                 dClusters,
                                                                                                 dMemberCount);
    }
}

template<typename ValueType, typename IndexType>
int CallfuncSync(cluster_kernel_t ct,std::string path) {
    ValueType *Data_cv;
    IndexType *Data_ptr;
    IndexType *Data_cid;
    int *Membership;
    int *MemberCount;
    ValueType *Clusters;
    bool Changed;

    int num_nz;
    get_nz<ValueType>(num_nz);
    if (num_nz <= 0) {
        std::cout << "Failed to read the data!" << std::endl;
        return -1;
    } else {
        std::cout << "get the number of non-zero elements";
    }

    checkCuda(hipHostAlloc((void **) &Data_cv, num_nz * sizeof(ValueType), hipHostMallocDefault));
    checkCuda(hipHostAlloc((void **) &Data_ptr, (kSrcCount + 1) * sizeof(IndexType), hipHostMallocDefault));
    checkCuda(hipHostAlloc((void **) &Data_cid, num_nz * sizeof(IndexType), hipHostMallocDefault));

    read_csr<ValueType, IndexType>(Data_cv, Data_ptr, Data_cid);

    checkCuda(hipHostAlloc((void **) &Clusters, kClusterCount * kCoords * sizeof(ValueType),
                            hipHostMallocDefault)); //FIXME: Initialize Cluster Center?
//    std::fill(Clusters, Clusters+kClusterCount*kCoords, 1); //FIXME:fill
    for (unsigned int i = 0; i < kClusterCount * kCoords; i++) { Clusters[i] = rand() % 10; }

    checkCuda(hipHostAlloc((void **) &Membership, kSrcCount * sizeof(int),
                            hipHostMallocDefault)); //use fixed host memory
    std::fill(Membership, Membership + kSrcCount, 0);

    checkCuda(hipHostAlloc((void **) &MemberCount, kClusterCount * sizeof(int), hipHostMallocDefault));
    std::fill(MemberCount, MemberCount + kClusterCount, 0);

    ValueType *dData_cv;
    IndexType *dData_ptr;
    IndexType *dData_cid;
    int *dMembership;
    int *dMemberCount;
    ValueType *dClusters;
    bool *dChanged;

    checkCuda(hipMalloc((void **) &dData_cv, num_nz * sizeof(ValueType)));
    checkCuda(hipMalloc((void **) &dData_ptr, (kSrcCount + 1) * sizeof(IndexType)));
    checkCuda(hipMalloc((void **) &dData_cid, num_nz * sizeof(IndexType)));

    checkCuda(hipMalloc((void **) &dClusters, kClusterCount * kCoords * sizeof(ValueType)));
    checkCuda(hipMalloc((void **) &dMembership, kSrcCount * sizeof(int)));
    checkCuda(hipMalloc((void **) &dMemberCount, kClusterCount * sizeof(int)));
    checkCuda(hipMalloc((void **) &dChanged, 1 * sizeof(bool)));

    checkCuda(hipMemcpy(dData_cv, Data_cv, num_nz * sizeof(ValueType), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(dData_ptr, Data_ptr, (kSrcCount + 1) * sizeof(IndexType), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(dData_cid, Data_cid, num_nz * sizeof(IndexType), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(dMembership, Membership, kSrcCount * sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(dMemberCount, MemberCount, kClusterCount * sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(dClusters, Clusters, kCoords * kClusterCount * sizeof(ValueType), hipMemcpyHostToDevice));

    ValueType *dDst;
    checkCuda(hipMalloc((void **) &dDst, kClusterCount * kSrcCount * sizeof(ValueType)));
    ValueType *Dst = new ValueType[kClusterCount * kSrcCount]{0};

    ValueType *dIntermediate;
    if (ct == MoreParallelReduction) {
        dim3 block_size(BLOCK_SIZE / 8, 8);
        dim3 grid_size((kSrcCount + block_size.x - 1) / block_size.x, kClusterCount);
        checkCuda(hipMalloc((void **) &dIntermediate, kClusterCount * grid_size.x * kCoords * sizeof(ValueType)));
    }

    int itCount = 0;
    do {
        /** update Memberships and MemberCount **/
        updateMembership<ValueType, IndexType, kCoords, kClusterCount, kSrcCount>(dDst, dMembership, dChanged, dData_cv,
                                                                                  dData_ptr, dData_cid, dClusters);
        checkCuda();
#if defined(DEBUG)
        checkCuda(hipMemcpy(Membership, dMembership, kSrcCount * sizeof(int), hipMemcpyDeviceToHost));
#endif

        checkCuda(hipMemcpy(&Changed, dChanged, sizeof(bool), hipMemcpyDeviceToHost));

        updateMemberCount<kClusterCount, kSrcCount>(dMembership, dMemberCount);
        checkCuda();
#if defined(DEBUG)
        checkCuda(hipMemcpy(MemberCount, dMemberCount, kClusterCount * sizeof(int), hipMemcpyDeviceToHost));
        printmat(MemberCount, 1, kClusterCount, __LINE__);
#endif

        /** update clusters **/
        if (ct == cpu) {
            checkCuda(hipMemcpy(Membership, dMembership, kSrcCount * sizeof(int), hipMemcpyDeviceToHost));
            updateClustersMemberCount_cpu<ValueType, IndexType, kCoords, kClusterCount, kSrcCount>(Clusters,
                                                                                                   MemberCount,
                                                                                                   Membership, Data_cv,
                                                                                                   Data_ptr,
                                                                                                   Data_cid);
            checkCuda();
            checkCuda(hipMemcpy(dClusters, Clusters, kCoords * kClusterCount * sizeof(ValueType),
                                 hipMemcpyHostToDevice));

        } else {
            updateClusters_cuda<ValueType, IndexType, kCoords, kClusterCount, kSrcCount>(ct, dClusters, dIntermediate,
                                                                                         dMembership, dMemberCount,
                                                                                         dData_cv, dData_ptr,
                                                                                         dData_cid);
#if defined(DEBUG)
            checkCuda(hipMemcpy(Clusters, dClusters, kCoords * kClusterCount * sizeof(ValueType),
                                 hipMemcpyDeviceToHost)); //TODO: only for debug
            printmat(Clusters, kClusterCount, kCoords, __LINE__);
#endif
        }
    } while ((Changed == true) && (itCount++ < loop_iteration));

    std::cout << "it count: " << itCount << std::endl;

    hipFree(dData_cv);
    hipFree(dData_ptr);
    hipFree(dData_cid);
    hipFree(dMembership);
    hipFree(dChanged);
    hipFree(dDst);

    hipHostFree(Data_cv);
    hipHostFree(Data_ptr);
    hipFree(Data_cid);
    hipHostFree(Membership);
    hipHostFree(MemberCount);
    hipHostFree(Clusters);

    return 0;
}

int main(){
    std::string path = "./../../data/blob/";
//    CallfuncSync<double,int>(cpu,path);
//    CallfuncSync<double,int>(naive,path);
    CallfuncSync<double,int>(naive,path);
}
